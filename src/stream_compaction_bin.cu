#include "hip/hip_runtime.h"
/*
  stream_compaction_bin.cu
  ---------------------------------------------------------------
  Bin-based stream compaction using Morton-code binning.
  Each bin is identified by the lower k bits of the Morton code.
  We build (binOffsets, binSizes) and compact every bin separately
  to improve shared-memory locality.

  Author : Kaixiang Zou
  Date   : 2025-07-14
*/

#include "stream_compaction_bin.h"
#include "stream_compaction.h"   // reuse compactNaiveGPU / compactSharedGPU
#include "benchmark_utils.h"
#include "bin_kernel.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/binary_search.h>       // for thrust::lower_bound / upper_bound
#include <thrust/adjacent_difference.h> // for thrust::adjacent_difference


#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#include <numeric>     // <-- add this

extern float d_threshold; 

/* -------------------------------------------------------------------------- */
/*  computeBinOffsets                                                         */
/* -------------------------------------------------------------------------- */
void computeBinOffsets(const uint32_t* d_codes,
                       int             N,
                       int             kBits,
                       int*            d_binOffsets,
                       int*            d_binSizes)
{
    /* 把 Morton 码拷到 host，做一次线性扫描 */
    std::vector<uint32_t> h_codes(N);
    hipMemcpy(h_codes.data(), d_codes, N * sizeof(uint32_t),
               hipMemcpyDeviceToHost);

    const int numBins  = 1 << kBits;
    std::vector<int> h_offsets(numBins + 1, 0);
    std::vector<int> h_sizes  (numBins,     0);

    int curBin = (h_codes.empty() ? 0 :
                  (h_codes[0] & ((1 << kBits) - 1)));

    for (int i = 0; i < N; ++i) {  
        int binID = h_codes[i] & ((1 << kBits) - 1);
        if (binID != curBin) {
            /* 记录上一个 bin 的终点 / 大小 */
            h_offsets[curBin + 1] = i;
            h_sizes[curBin]       = i - h_offsets[curBin];
            /* 填充可能空缺的 bin（如果数据稀疏） */
            for (int b = curBin + 1; b < binID; ++b) {
                h_offsets[b + 1] = i;
                h_sizes[b]       = 0;
            }
            curBin = binID;
            h_offsets[curBin] = i;   // 起点
        }
    }
    /* 最后一个 bin */
    h_offsets[numBins]           = N;
    h_sizes[curBin]              = N - h_offsets[curBin];

    /* 拷回 device */
    hipMemcpy(d_binOffsets, h_offsets.data(),
               (numBins + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_binSizes,   h_sizes.data(),
               numBins      * sizeof(int), hipMemcpyHostToDevice);
}


/* -------------------------------------------------------------------------- */
/*  compactWithBinsGPU  – compact each bin on the GPU                         */
/* -------------------------------------------------------------------------- */
void compactWithBinsGPU(const Point2D*  d_in,
                        Point2D*        d_out,
                        const uint32_t* mortonCodes,
                        int             numPoints,
                        int             kBits,
                        int*            d_outCount)
{
    const int numBins = 1 << kBits;

    /* --- build bin metadata on device --- */
    thrust::device_vector<int> d_offsets(numBins + 1);
    thrust::device_vector<int> d_sizes  (numBins);

    computeBinOffsets(mortonCodes, numPoints, kBits,
                      thrust::raw_pointer_cast(d_offsets.data()),
                      thrust::raw_pointer_cast(d_sizes.data()));

    /* --- copy to host for a simple loop --- */
    std::vector<int> h_offsets(numBins + 1);
    std::vector<int> h_sizes  (numBins);
    thrust::copy(d_offsets.begin(), d_offsets.end(), h_offsets.begin());
    thrust::copy(d_sizes.begin(),   d_sizes.end(),   h_sizes.begin());

    int totalCompacted = 0;

    for (int bin = 0; bin < numBins; ++bin) {
        int offsetIn = h_offsets[bin];
        int sizeIn   = h_sizes[bin];
        if (sizeIn == 0) continue;            // skip empty bin

        const Point2D* binIn  = d_in  + offsetIn;
        Point2D*       binOut = d_out + totalCompacted;

        int h_count = 0;                      // host-side counter
        compactNaiveGPU(binIn, binOut, sizeIn, h_count);

        totalCompacted += h_count;
    }

    /* --- final total back to device --- */
    hipMemcpy(d_outCount, &totalCompacted,
               sizeof(int), hipMemcpyHostToDevice);
}



/* -------------------------------------------------------------------------- */
/*  runBitmaskBenchmarkWithBins                                               */
/* -------------------------------------------------------------------------- */
void runBitmaskBenchmarkWithBins(int               size,
                                 int               blockSize,
                                 const std::string precision,
                                 float&            time_ms,
                                 float&            error)
{
    // Placeholder: integrate with your existing benchmark utilities.
    // Steps you’ll likely need:
    //   1. Allocate / generate input data (points + Morton codes)
    //   2. hipEvent_t start/stop around compactWithBinsGPU()
    //   3. Compute error vs. CPU reference if desired
    //   4. Release resources
    std::cout << "[bin-mode] benchmark stub (size = " << size
              << ", block = " << blockSize
              << ", precision = " << precision << ")\n";
    time_ms = 0.0f;
    error   = 0.0f;
}

/* -------------------------------------------------------------------------- */
/*  host wrapper similar to testNaiveGPUCompaction                            */
/* -------------------------------------------------------------------------- */
void testBinGPUCompaction(const std::vector<Point2D>& input,
                          float                       threshold,
                          int                         kBits,
                          std::vector<Point2D>&       output) {
    const int N = static_cast<int>(input.size());

    // 1. allocate and copy input
    Point2D* d_in  = nullptr;
    Point2D* d_out = nullptr;
    uint32_t* d_codes = nullptr;         // Morton codes on device
    int* d_outCount   = nullptr;

    hipMalloc(&d_in,  N * sizeof(Point2D));
    hipMalloc(&d_out, N * sizeof(Point2D));
    hipMalloc(&d_codes, N * sizeof(uint32_t));
    hipMalloc(&d_outCount, sizeof(int));

    hipMemcpy(d_in, input.data(), N * sizeof(Point2D), hipMemcpyHostToDevice);

    // 1.1 prepare Morton codes (host → compute & copy，或在 device 上 kernel 计算)
    std::vector<uint32_t> codes(N);
    for (int i = 0; i < N; ++i) codes[i] = morton2D_encode((int)input[i].x, (int)input[i].y);
    hipMemcpy(d_codes, codes.data(), N * sizeof(uint32_t), hipMemcpyHostToDevice);

    // 2. call bin compaction
    compactWithBinsGPU(d_in, d_out, d_codes, N, kBits, d_outCount);

    // 3. copy back result
    int h_outCount = 0;
    hipMemcpy(&h_outCount, d_outCount, sizeof(int), hipMemcpyDeviceToHost);

    output.resize(h_outCount);
    hipMemcpy(output.data(), d_out, h_outCount * sizeof(Point2D), hipMemcpyDeviceToHost);

    // 4. free
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_codes);
    hipFree(d_outCount);
}


/* -------------------------------------------------------------------------- */
/*  compactBinAtomic  ——  one-pass compaction with atomics                    */
/* -------------------------------------------------------------------------- */
// __global__ void compactBinAtomic(const Point2D* __restrict__ in,
//                                  Point2D*       __restrict__ out,
//                                  const uint32_t*__restrict__ codes,
//                                  int*           binCursor,
//                                  int            N,
//                                  int            mask,
//                                  float          threshold)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= N) return;

//     Point2D p = in[idx];
//     if (p.temp <= threshold) return;            // filter condition

//     int binId = codes[idx] & mask;              // low-k bits
//     int pos   = atomicAdd(&binCursor[binId], 1);
//     out[pos]  = p;                              // write directly
// }

// __global__ void compactBinAtomic(const Point2D*  in,
//                                  Point2D*        out,
//                                  int*            binCursor,
//                                  const int*      binOffsets,
//                                  const uint32_t* codes,
//                                  int             N,
//                                  int             mask,
//                                  float           thr)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= N) return;

//     Point2D p = in[idx];
//     if (p.temp > thr) {
//         int bin       = codes[idx] & mask;
//         int posInBin  = atomicAdd(&binCursor[bin], 1);
//         int globalPos = binOffsets[bin] + posInBin;
//         out[globalPos] = p;
//     }
// }

/* ---------------------------------------------------------------------------
   compactBinAtomic  –  Plan-B, one-pass atomic compaction
   把满足谓词的点直接写到输出，全局递增计数器保证写入唯一
--------------------------------------------------------------------------- */
__global__ void compactBinAtomic(const Point2D*  in,
                                 Point2D*        out,
                                 int*            globalCnt,     // 唯一计数器
                                 const uint32_t* mortonCodes,
                                 int             N,
                                 int             mask,          // 低 k 位 (未用，可保留)
                                 float           thr)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    Point2D p = in[idx];
    if (p.temp > thr) {                         // 谓词：温度 > threshold
        int pos = atomicAdd(globalCnt, 1);      // 获得全局写位置
        out[pos] = p;                           // 写入
    }
}




// void testBinGPUCompaction_atomic(const std::vector<Point2D>& input,
//                                  float                       threshold,
//                                  int                         kBits,
//                                  std::vector<Point2D>&       output,
//                                  float&                      t_kernel_ms,
//                                  float&                      t_total_ms)
// {
//     const int N       = static_cast<int>(input.size());
//     const int numBins = 1 << kBits;
//     const int mask    = numBins - 1;

//     /* ---------- CUDA events ---------- */
//     hipEvent_t t0, t1, k0, k1;
//     hipEventCreate(&t0); hipEventCreate(&t1);
//     hipEventCreate(&k0); hipEventCreate(&k1);
//     hipEventRecord(t0);

//     /* ---------- raw buffers ---------- */
//     Point2D*  d_in  = nullptr;
//     Point2D*  d_out = nullptr;
//     uint32_t* d_codes = nullptr;
//     hipMalloc(&d_in,  N * sizeof(Point2D));
//     hipMalloc(&d_out, N * sizeof(Point2D));
//     hipMalloc(&d_codes, N * sizeof(uint32_t));
//     hipMemcpy(d_in, input.data(), N*sizeof(Point2D), hipMemcpyHostToDevice);

//     /* ---------- device vectors for Plan-B ---------- */
//     thrust::device_vector<int> d_binSizes  (numBins,   0);
//     thrust::device_vector<int> d_binOffsets(numBins+1, 0);
//     thrust::device_vector<int> d_binCursor (numBins,   0);   // atomic counter

//     /* ---------- build Morton codes on host ---------- */
//     std::vector<uint32_t> h_codes(N);
//     for (int i = 0; i < N; ++i)
//         h_codes[i] = morton2D_encode(static_cast<int>(input[i].x),
//                                      static_cast<int>(input[i].y));
//     hipMemcpy(d_codes, h_codes.data(), N*sizeof(uint32_t),
//                hipMemcpyHostToDevice);

//     /* ---------- pass-1 histogram ---------- */
//     const int threads = 256;
//     const int blocks  = (N + threads - 1) / threads;
//     histogramBins<<<blocks,threads>>>(d_codes,
//         thrust::raw_pointer_cast(d_binSizes.data()),
//         N, mask);

//     /* ---------- exclusive scan → offsets ---------- */
//     thrust::exclusive_scan(d_binSizes.begin(), d_binSizes.end(),
//                            d_binOffsets.begin());
//     d_binOffsets[numBins] = N;   // last offset = N

//     /* ---------- kernel launch (atomic Plan-B) ------ */
//     hipEventRecord(k0);
//     compactBinAtomic<<<blocks,threads>>>(
//         d_in, d_out,
//         thrust::raw_pointer_cast(d_binCursor.data()),
//         thrust::raw_pointer_cast(d_binOffsets.data()),
//         d_codes, N, mask, threshold);
//     hipEventRecord(k1);
//     hipEventSynchronize(k1);

//     /* ---------- gather counts & copy back ---------- */
//     std::vector<int> h_cursor(numBins);
//     hipMemcpy(h_cursor.data(), thrust::raw_pointer_cast(d_binCursor.data()),
//                numBins*sizeof(int), hipMemcpyDeviceToHost);
//     int total = std::accumulate(h_cursor.begin(), h_cursor.end(), 0);

//     output.resize(total);
//     hipMemcpy(output.data(), d_out,
//                total*sizeof(Point2D), hipMemcpyDeviceToHost);

//     hipEventRecord(t1); hipEventSynchronize(t1);
//     hipEventElapsedTime(&t_kernel_ms, k0, k1);
//     hipEventElapsedTime(&t_total_ms,  t0, t1);

//     /* ---------- cleanup ---------- */
//     hipEventDestroy(k0); hipEventDestroy(k1);
//     hipEventDestroy(t0); hipEventDestroy(t1);
//     hipFree(d_in); hipFree(d_out); hipFree(d_codes);
// }

/* --------------------------------------------------------------------------
   testBinGPUCompaction_atomic  –  Plan-B host driver
   不再做直方图 / binOffsets，直接使用全局计数器
--------------------------------------------------------------------------- */
void testBinGPUCompaction_atomic(const std::vector<Point2D>& input,
                                 float                       threshold,
                                 int                         kBits,
                                 std::vector<Point2D>&       output,
                                 float&                      t_kernel_ms,
                                 float&                      t_total_ms)
{
    const int N = static_cast<int>(input.size());

    /* ---------- CUDA events ---------- */
    hipEvent_t t0, t1, k0, k1;
    hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventCreate(&k0); hipEventCreate(&k1);
    hipEventRecord(t0);

    /* ---------- allocate buffers ---------- */
    Point2D*  d_in  = nullptr;
    Point2D*  d_out = nullptr;
    uint32_t* d_codes = nullptr;
    int*      d_globalCnt = nullptr;          // 全局计数器

    hipMalloc(&d_in,  N * sizeof(Point2D));
    hipMalloc(&d_out, N * sizeof(Point2D));
    hipMalloc(&d_codes, N * sizeof(uint32_t));
    hipMalloc(&d_globalCnt, sizeof(int));
    hipMemset(d_globalCnt, 0, sizeof(int));  // 计数器清零

    /* ---------- copy input points ---------- */
    hipMemcpy(d_in, input.data(),
               N * sizeof(Point2D), hipMemcpyHostToDevice);

    /* ---------- build Morton codes on host ---------- */
    std::vector<uint32_t> h_codes(N);
    for (int i = 0; i < N; ++i)
        h_codes[i] = morton2D_encode(
                        static_cast<int>(input[i].x),
                        static_cast<int>(input[i].y));
    hipMemcpy(d_codes, h_codes.data(),
               N * sizeof(uint32_t), hipMemcpyHostToDevice);

    /* ---------- launch kernel ---------- */
    const int threads = 256;
    const int blocks  = (N + threads - 1) / threads;
    hipEventRecord(k0);

    compactBinAtomic<<<blocks, threads>>>(d_in, d_out,
                                          d_globalCnt,
                                          d_codes, N,
                                          /*mask*/ (1 << kBits) - 1,
                                          threshold);

    hipEventRecord(k1);
    hipEventSynchronize(k1);

    /* ---------- copy result count ---------- */
    int total = 0;
    hipMemcpy(&total, d_globalCnt,
               sizeof(int), hipMemcpyDeviceToHost);

    /* ---------- copy compacted points back ---------- */
    output.resize(total);
    if (total > 0) {
        hipMemcpy(output.data(), d_out,
                   total * sizeof(Point2D), hipMemcpyDeviceToHost);
    }

    /* ---------- timing ---------- */
    hipEventRecord(t1);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t_kernel_ms, k0, k1);
    hipEventElapsedTime(&t_total_ms,  t0, t1);

    /* ---------- cleanup ---------- */
    hipEventDestroy(k0);  hipEventDestroy(k1);
    hipEventDestroy(t0);  hipEventDestroy(t1);
    hipFree(d_in); hipFree(d_out);
    hipFree(d_codes); hipFree(d_globalCnt);
}



/* pass-1: build histogram (binSizes) */
__global__ void histogramBins(const uint32_t* codes,
                              int*           binSizes,
                              int            N,
                              int            mask)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    int id = codes[idx] & mask;
    atomicAdd(&binSizes[id], 1);
}

/* pass-2: scatter points into tmp so each bin is contiguous */
__global__ void scatterToBins(const Point2D* __restrict__ in,
                              Point2D*       __restrict__ tmp,
                              const uint32_t*__restrict__ codes,
                              int*           binCursor,   // init = binOffsets
                              int            N,
                              int            mask)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    int id  = codes[idx] & mask;
    int pos = atomicAdd(&binCursor[id], 1);   // unique slot in that bin
    tmp[pos] = in[idx];
}

/* -------------------------------------------------------------------------- */
/*  testBinGPUCompaction_partition  (Plan-A)                                  */
/* -------------------------------------------------------------------------- */
void testBinGPUCompaction_partition(const std::vector<Point2D>& input,
                                    float                       threshold,
                                    int                         kBits,
                                    std::vector<Point2D>&       output,
                                    float&                      t_kernel_ms,
                                    float&                      t_total_ms,
                                    BinKernel kernelKind)
{
    const int N       = static_cast<int>(input.size());
    const int numBins = 1 << kBits;
    const int mask    = numBins - 1;

    /* ---------- CUDA events ---------- */
    hipEvent_t t0, t1, k0, k1;
    hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventCreate(&k0); hipEventCreate(&k1);
    hipEventRecord(t0);

    /* ---------- allocate & H2D copy ---------- */
    Point2D*  d_in;      hipMalloc(&d_in,  N * sizeof(Point2D));
    Point2D*  d_tmp;     hipMalloc(&d_tmp, N * sizeof(Point2D));   // scatter buffer
    Point2D*  d_out;     hipMalloc(&d_out, N * sizeof(Point2D));
    uint32_t* d_codes;   hipMalloc(&d_codes, N * sizeof(uint32_t));

    hipMemcpy(d_in, input.data(), N*sizeof(Point2D), hipMemcpyHostToDevice);

    thrust::device_vector<int>  d_binSizes (numBins, 0);
    thrust::device_vector<int>  d_binOffsets(numBins+1, 0);

    /* ---------- prepare Morton codes ---------- */
    std::vector<uint32_t> h_codes(N);
    for (int i=0;i<N;++i)
        h_codes[i] = morton2D_encode((int)input[i].x, (int)input[i].y);
    hipMemcpy(d_codes, h_codes.data(), N*sizeof(uint32_t), hipMemcpyHostToDevice);

    /* ---------- pass-1 histogram ---------- */
    int threads = 256, blocks = (N+threads-1)/threads;
    histogramBins<<<blocks,threads>>>(d_codes,
                                      thrust::raw_pointer_cast(d_binSizes.data()),
                                      N, mask);
    /* ---------- exclusive scan ---------- */
    thrust::exclusive_scan(d_binSizes.begin(), d_binSizes.end(),
                           d_binOffsets.begin());
    // 把最后一个 offset 设为 N
    d_binOffsets[numBins] = N;

    /* ---------- pass-2 scatter ---------- */
    // binCursor = binOffsets (拷贝一份)
    thrust::device_vector<int> d_binCursor = d_binOffsets;
    hipEventRecord(k0);         // kernel timer start
    scatterToBins<<<blocks,threads>>>(d_in, d_tmp, d_codes,
                                      thrust::raw_pointer_cast(d_binCursor.data()),
                                      N, mask);
    hipEventRecord(k1);         // kernel timer end
    hipEventSynchronize(k1);

    /* ---------- copy offsets & sizes to host ---------- */
    std::vector<int> h_offsets(numBins+1);
    std::vector<int> h_sizes  (numBins);
    thrust::copy(d_binOffsets.begin(), d_binOffsets.end(), h_offsets.begin());
    thrust::copy(d_binSizes.begin(),   d_binSizes.end(),   h_sizes.begin());

    /* ---------- per-bin compaction (shared / warp) ---------- */
    int totalOut = 0;
    for (int b=0;b<numBins;++b) {
        int off = h_offsets[b];
        int sz  = h_sizes[b];
        if (sz==0) continue;

        Point2D* binIn  = d_tmp + off;
        Point2D* binOut = d_out + totalOut;
        int      h_cnt  = 0;
        // TODO: replace with your optimized kernel
        //compactNaiveGPU(binIn, binOut, sz, h_cnt, threshold);
        //compactNaiveGPU(binIn, binOut, sz, h_cnt);
        //compactSharedGPU(binIn, binOut, sz, threshold, h_cnt); 
        compactOneBin(binIn, binOut, sz, threshold, h_cnt, kernelKind);
        
        totalOut += h_cnt;
    }

    /* ---------- copy results back ---------- */
    output.resize(totalOut);
    hipMemcpy(output.data(), d_out,
               totalOut*sizeof(Point2D), hipMemcpyDeviceToHost);

    hipEventRecord(t1); hipEventSynchronize(t1);
    hipEventElapsedTime(&t_kernel_ms, k0, k1);
    hipEventElapsedTime(&t_total_ms,  t0, t1);

    /* ---------- cleanup ---------- */
    hipFree(d_in); hipFree(d_tmp); hipFree(d_out); hipFree(d_codes);
    hipEventDestroy(k0); hipEventDestroy(k1);
    hipEventDestroy(t0); hipEventDestroy(t1);
}

// -----------------------------------------------------------------------------
//  compactWarpGPU – per-bin warp-shuffle stream compaction
//  每个 bin 使用 warp-shuffle 版本做压缩
// -----------------------------------------------------------------------------
void compactWarpGPU(const Point2D* d_in,
                    Point2D*       d_out,
                    int            N,
                    float          threshold,
                    int&           h_outCount)
{
    /* 0. Push predicate threshold to constant memory
       0. 把阈值写入 device constant memory */
    hipMemcpyToSymbol(HIP_SYMBOL(d_threshold), &threshold, sizeof(float));

    /* 1. Device counter for warp kernel
       1. 为 warp kernel 分配设备端计数器 */
    int* d_cnt = nullptr;
    hipMalloc(&d_cnt, sizeof(int));
    hipMemset(d_cnt, 0, sizeof(int));

    /* 2. Launch warp-shuffle compaction
       2. 启动 warp-shuffle 压缩 kernel */
    compact_points_warp(const_cast<Point2D*>(d_in), d_out, d_cnt, N);

    /* 3. Copy compacted count back
       3. 拷回压缩后元素个数 */
    hipMemcpy(&h_outCount, d_cnt, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_cnt);
}


/* ---------------------------------------------------------------------------
   compactOneBin – unified per-bin launcher
   根据指定策略调用 Shared / Warp / Bitmask kernel
--------------------------------------------------------------------------- */
void compactOneBin(Point2D* d_in,      // bin input (contiguous)
                   Point2D* d_out,     // bin output base
                   int      N,         // elements in this bin
                   float    threshold, // predicate value
                   int&     h_outCnt,  // host-side result count
                   BinKernel kind)     // strategy
{
    /* ---------------- threshold 常量区 ---------------- */
    hipMemcpyToSymbol(HIP_SYMBOL(d_threshold), &threshold, sizeof(float));

    /* ---------------- device counter ----------------- */
    int* d_cnt = nullptr;
    hipMalloc(&d_cnt, sizeof(int));
    hipMemset(d_cnt, 0, sizeof(int));

    /* ---------------- strategy select ---------------- */
    if (kind == BinKernel::Auto)        // simple heuristic
        kind = (N <= 32)   ? BinKernel::Bitmask :
               (N <= 1024) ? BinKernel::Warp
                            : BinKernel::Shared;

    switch (kind) {
      case BinKernel::Shared:
        compactSharedGPU(d_in, d_out, N, threshold, h_outCnt);
        break;

      case BinKernel::Warp:
        // kernel expects device counter ptr
        compact_points_warp(d_in, d_out, d_cnt, N);
        hipMemcpy(&h_outCnt, d_cnt, sizeof(int), hipMemcpyDeviceToHost);
        break;

      case BinKernel::Bitmask:
        compact_points_bitmask(d_in, d_out, d_cnt, N);
        hipMemcpy(&h_outCnt, d_cnt, sizeof(int), hipMemcpyDeviceToHost);
        break;

      default: break;
    }
    hipFree(d_cnt);
}